#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator
   https://www.lammps.org/, Sandia National Laboratories
   LAMMPS development team: developers@lammps.org

   Copyright (2003) Sandia Corporation.  Under the terms of Contract
   DE-AC04-94AL85000 with Sandia Corporation, the U.S. Government retains
   certain rights in this software.  This software is distributed under
   the GNU General Public License.

   See the README file in the top-level LAMMPS directory.
------------------------------------------------------------------------- */

/* ----------------------------------------------------------------------
   Contributing author: Gijin Kim, Hyungmin An (SNU)
------------------------------------------------------------------------- */

#include "pair_d3.h"

using namespace LAMMPS_NS;

/* --------- Macros for CUDA error handling --------- */
#define START_CUDA_TIMER()    \
    hipEvent_t start, stop;  \
    hipEventCreate(&start);  \
    hipEventCreate(&stop);   \
    hipEventRecord(start);

#define STOP_CUDA_TIMER(tag)                           \
    hipEventRecord(stop);                             \
    hipEventSynchronize(stop);                        \
    float msec = 0;                                    \
    hipEventElapsedTime(&msec, start, stop);          \
    printf("Elapsed time for %s: %f ms\n", tag, msec); \
    hipEventDestroy(start);                           \
    hipEventDestroy(stop);

#define CHECK_CUDA(call) do {                                            \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA Error (%s:%d) -> %s: %s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
} while (0)

#define CHECK_CUDA_ERROR() do {                                          \
    hipDeviceSynchronize();                                             \
    hipError_t status_ = hipGetLastError();                            \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA Error (%s:%d) -> %s: %s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
} while (0)

#define CHECK_CUDA_DEVICES() do {                                              \
    int deviceCount = 0;                                                       \
    if (hipGetDeviceCount(&deviceCount) != hipSuccess || deviceCount == 0) { \
        fprintf(stderr, "CUDA Error (%s:%d) -> No CUDA devices found\n",         \
                __FILE__, __LINE__);                                           \
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
} while(0)
/* --------- Macros for CUDA error handling --------- */

/* --------- Math functions for CUDA compatibility --------- */
inline __host__ __device__ void ij_at_linij(int linij, int &i, int &j) {
    i = static_cast<int>((sqrt(1 + 8 * linij) - 1) / 2);
    j = linij - i * (i + 1) / 2;
} // unroll the triangular loop

inline __host__ __device__ float lensq3(const float *v)
{
  return v[0] * v[0] + v[1] * v[1] + v[2] * v[2];
} // from MathExtra::lensq3
/* --------- Math functions for CUDA compatibility --------- */

/* ----------------------------------------------------------------------
   Constructor (Required)
------------------------------------------------------------------------- */

PairD3::PairD3(LAMMPS* lmp) : Pair(lmp) {
    single_enable = 0;      // potential is not pair-wise additive.
    restartinfo = 0;        // Many-body potentials are usually not
                            // written to binary restart files.
    one_coeff = 1;          // Many-body potnetials typically read all
                            // parameters from a file, so only one
                            // pair_coeff statement is needed.
    manybody_flag = 1;
    no_virial_fdotr_compute = 1;
}

/* ----------------------------------------------------------------------
   Destructor (Required)
------------------------------------------------------------------------- */

PairD3::~PairD3() {
    if (allocated) {
        int n = atom->natoms;
        int np1 = atom->ntypes + 1;
        int vdw_range_x = 2 * rep_vdw[0] + 1;
        int vdw_range_y = 2 * rep_vdw[1] + 1;
        int vdw_range_z = 2 * rep_vdw[2] + 1;
        int cn_range_x  = 2 * rep_cn[0] + 1;
        int cn_range_y  = 2 * rep_cn[1] + 1;
        int cn_range_z  = 2 * rep_cn[2] + 1;

        for (int i = 0; i < np1; i++) { hipFree(setflag[i]); }; hipFree(setflag);
        for (int i = 0; i < np1; i++) { hipFree(cutsq[i]); }; hipFree(cutsq);
        hipFree(r2r4);
        hipFree(rcov);
        hipFree(mxc);
        for (int i = 0; i < np1; i++) { hipFree(r0ab[i]); }; hipFree(r0ab);
        for (int i = 0; i < np1; i++) {
            for (int j = 0; j < np1; j++) {
                for (int k = 0; k < MAXC; k++) {
                    for (int l = 0; l < MAXC; l++) {
                        hipFree(c6ab[i][j][k][l]);
                    }
                    hipFree(c6ab[i][j][k]);
                }
                hipFree(c6ab[i][j]);
            }
            hipFree(c6ab[i]);
        }
        hipFree(c6ab);

        hipFree(lat_v_1);
        hipFree(lat_v_2);
        hipFree(lat_v_3);

        hipFree(rep_vdw);
        hipFree(rep_cn);
        hipFree(cn);
        for (int i = 0; i < n; i++) { hipFree(x[i]); }; hipFree(x);

        hipFree(dc6i);
        for (int i = 0; i < n; i++) { hipFree(f[i]); }; hipFree(f);

        for (int i = 0; i < 3; i++) { hipFree(sigma[i]); }; hipFree(sigma);

        hipFree(dc6_iji_tot);
        hipFree(dc6_ijj_tot);
        hipFree(c6_ij_tot);

        for (int i = 0; i < vdw_range_x; i++) {
            for (int j = 0; j < vdw_range_y; j++) {
                for (int k = 0; k < vdw_range_z; k++) {
                    hipFree(tau_vdw[i][j][k]);
                }
                hipFree(tau_vdw[i][j]);
            }
            hipFree(tau_vdw[i]);
        }
        hipFree(tau_vdw);
        for (int i = 0; i < cn_range_x; i++) {
            for (int j = 0; j < cn_range_y; j++) {
                for (int k = 0; k < cn_range_z; k++) {
                    hipFree(tau_cn[i][j][k]);
                }
                hipFree(tau_cn[i][j]);
            }
            hipFree(tau_cn[i]);
        }
        hipFree(tau_cn);

        hipFree(tau_idx_vdw);
        hipFree(tau_idx_cn);

        hipFree(atomtype);
        hipFree(disp);
    }
}

/* ----------------------------------------------------------------------
   Allocate all arrays (Required)
------------------------------------------------------------------------- */

void PairD3::allocate() {
    CHECK_CUDA_DEVICES();
    allocated = 1;

    /* atom->ntypes : # of elements; element index starts from 1 */
    int n = atom->natoms;
    int np1 = atom->ntypes + 1;
    n_save = n;

    hipMallocManaged(&setflag, np1 * sizeof(int*)); for (int i = 0; i < np1; i++) { hipMallocManaged(&setflag[i], np1 * sizeof(int)); }
    hipMallocManaged(&cutsq, np1 * sizeof(double*)); for (int i = 0; i < np1; i++) { hipMallocManaged(&cutsq[i], np1 * sizeof(double)); }
    hipMallocManaged(&r2r4, np1 * sizeof(float));
    hipMallocManaged(&rcov, np1 * sizeof(float));
    hipMallocManaged(&mxc, np1 * sizeof(int));
    hipMallocManaged(&r0ab, np1 * sizeof(float*)); for (int i = 0; i < np1; i++) { hipMallocManaged(&r0ab[i], np1 * sizeof(float)); }
    hipMallocManaged(&c6ab, np1 * sizeof(float****));
    for (int i = 0; i < np1; i++) {
        hipMallocManaged(&c6ab[i], np1 * sizeof(float***));
        for (int j = 0; j < np1; j++) {
            hipMallocManaged(&c6ab[i][j], MAXC * sizeof(float**));
            for (int k = 0; k < MAXC; k++) {
                hipMallocManaged(&c6ab[i][j][k], MAXC * sizeof(float*));
                for (int l = 0; l < MAXC; l++) {
                    hipMallocManaged(&c6ab[i][j][k][l], 3 * sizeof(float));
                }
            }
        }
    }

    hipMallocManaged(&lat_v_1, 3 * sizeof(float));
    hipMallocManaged(&lat_v_2, 3 * sizeof(float));
    hipMallocManaged(&lat_v_3, 3 * sizeof(float));
    hipMallocManaged(&rep_vdw, 3 * sizeof(int));
    hipMallocManaged(&rep_cn,  3 * sizeof(int));
    hipMallocManaged(&sigma,   3 * sizeof(double*)); for (int i = 0; i < 3; i++) { hipMallocManaged(&sigma[i], 3 * sizeof(double)); }

    hipMallocManaged(&cn, n * sizeof(double));
    hipMallocManaged(&x, n * sizeof(float*)); for (int i = 0; i < n; i++) { hipMallocManaged(&x[i], 3 * sizeof(float)); }
    hipMallocManaged(&dc6i, n * sizeof(double));
    hipMallocManaged(&f, n * sizeof(double*)); for (int i = 0; i < n; i++) { hipMallocManaged(&f[i], 3 * sizeof(double)); }

    // Initialization
    // Initialize for lattice -> set_lattice_vectors()
    tau_idx_vdw_total_size = -1;
    tau_idx_cn_total_size = -1;
    for (int i = 0; i < 3; i++) {
        rep_vdw[i] = -1;
        rep_cn[i] = -1;
    }

    for (int i = 1; i < np1; i++) {
        for (int j = 1; j < np1; j++) {
            setflag[i][j] = 0;
        }
    }

    for (int idx1 = 0; idx1 < np1; idx1++) {
        for (int idx2 = 0; idx2 < np1; idx2++) {
            for (int idx3 = 0; idx3 < MAXC; idx3++) {
                for (int idx4 = 0; idx4 < MAXC; idx4++) {
                    for (int idx5 = 0; idx5 < 3; idx5++) {
                        c6ab[idx1][idx2][idx3][idx4][idx5] = -1;
                    }
                }
            }
        }
    }

    int n_ij_combination = n * (n + 1) / 2;
    hipMallocManaged(&dc6_iji_tot, n_ij_combination * sizeof(float));
    hipMallocManaged(&dc6_ijj_tot, n_ij_combination * sizeof(float));
    hipMallocManaged(&c6_ij_tot,   n_ij_combination * sizeof(float));

    hipMallocManaged(&atomtype, n * sizeof(int));
    hipMallocManaged(&disp, sizeof(double));
}

/* ----------------------------------------------------------------------
   Settings : read from pair_style (Required) -> pair_style d3 vdw_sq cn_sq damp_name func_name
------------------------------------------------------------------------- */

void PairD3::settings(int narg, char **arg) {
    if (narg != 4) {
        error->all(FLERR,
                "Pair_style d3 needs Four arguments:\n"
                "\t rthr: cutoff radius for dispersion interaction (a.u.^2)\n"
                "\t cnthr: cutoff raius for coordination number (a.u.^2)\n"
                "\t damping: name of the damping function (e.g., damp_zero, damp_bj)\n"
                "\t functional: name of the functional (e.g., pbe, b3-lyp)\n"
                );
    }
    rthr   = utils::numeric(FLERR, arg[0], false, lmp);
    cnthr = utils::numeric(FLERR, arg[1], false, lmp);

    std::map<std::string, int> commandMap = {
        {"damp_zero", 0}, {"damp_bj", 1}, {"damp_zerom", 2}, {"damp_bjm", 3},
    };

    if (commandMap.find(arg[2]) == commandMap.end()) {
        error->all(FLERR, "Unknown damping function");
    }

    damping = commandMap[arg[2]];
    functional = arg[3];

    setfuncpar();
}

/* ----------------------------------------------------------------------
   finds atomic number (used in PairD3::coeff)
------------------------------------------------------------------------- */

int PairD3::find_atomic_number(std::string& key) {
    std::transform(key.begin(), key.end(), key.begin(), ::tolower);
    if (key.length() == 1) { key += " "; }
    key.resize(2);

    std::vector<std::string> element_table = {
        "h ","he",
        "li","be","b ","c ","n ","o ","f ","ne",
        "na","mg","al","si","p ","s ","cl","ar",
        "k ","ca","sc","ti","v ","cr","mn","fe","co","ni","cu",
        "zn","ga","ge","as","se","br","kr",
        "rb","sr","y ","zr","nb","mo","tc","ru","rh","pd","ag",
        "cd","in","sn","sb","te","i ","xe",
        "cs","ba","la","ce","pr","nd","pm","sm","eu","gd","tb","dy",
        "ho","er","tm","yb","lu","hf","ta","w ","re","os","ir","pt",
        "au","hg","tl","pb","bi","po","at","rn",
        "fr","ra","ac","th","pa","u ","np","pu"
    };

    for (size_t i = 0; i < element_table.size(); ++i) {
        if (element_table[i] == key) {
            int atomic_number = i + 1;
            return atomic_number;
        }
    }

    // if not the case
    return -1;
}

/* ----------------------------------------------------------------------
   Check whether an integer value in an integer array (used in PairD3::coeff)
------------------------------------------------------------------------- */

int PairD3::is_int_in_array(int arr[], int size, int value) {
    for (int i = 0; i < size; i++) {
        if (arr[i] == value) { return i; } // returns the index
    }
    return -1;
}

/* ----------------------------------------------------------------------
   Read r0ab values from the table (used in PairD3::coeff)
------------------------------------------------------------------------- */

void PairD3::read_r0ab(int* atomic_numbers, int ntypes) {
    const double r0ab_table[94][94] = R0AB_TABLE;

    for (int i = 1; i <= ntypes; i++) {
        for (int j = 1; j <= ntypes; j++) {
            r0ab[i][j] = r0ab_table[atomic_numbers[i-1]-1][atomic_numbers[j-1]-1] / AU_TO_ANG;
        }
    }
}

/* ----------------------------------------------------------------------
   Get atom pair indices and grid indices (used in PairD3::read_c6ab)
------------------------------------------------------------------------- */

void PairD3::get_limit_in_pars_array(int& idx_atom_1, int& idx_atom_2, int& idx_i, int& idx_j) {
    const int shift = 100;

    idx_i = (idx_atom_1 - 1) / shift + 1;
    idx_j = (idx_atom_2 - 1) / shift + 1;

    idx_atom_1 = (idx_atom_1 - 1) % shift + 1;
    idx_atom_2 = (idx_atom_2 - 1) % shift + 1;

    // the code above replaces the code below
    //idx_i = 1;
    //idx_j = 1;
    //int shift = 100;
    //while (idx_atom_1 > shift) { idx_atom_1 -= shift; idx_i++; }
    //while (idx_atom_2 > shift) { idx_atom_2 -= shift; idx_j++; }
}

/* ----------------------------------------------------------------------
   Read c6ab values from the table (used in PairD3::coeff)
------------------------------------------------------------------------- */

void PairD3::read_c6ab(int* atomic_numbers, int ntypes) {
    for (int i = 1; i <= ntypes; i++) { mxc[i] = 0; }
    int grid_i = 0, grid_j = 0;

    const double c6ab_table[32385][5] = C6AB_TABLE;

    for (int i = 0; i < 32385; i++) {
        const double ref_c6 = c6ab_table[i][0];
        int atom_number_1 = static_cast<int>(c6ab_table[i][1]);
        int atom_number_2 = static_cast<int>(c6ab_table[i][2]);
        get_limit_in_pars_array(atom_number_1, atom_number_2, grid_i, grid_j);
        const int idx_atom_1 = is_int_in_array(atomic_numbers, ntypes, atom_number_1);
        if (idx_atom_1 < 0) { continue; }
        const int idx_atom_2 = is_int_in_array(atomic_numbers, ntypes, atom_number_2);
        if (idx_atom_2 < 0) { continue; }
        const double ref_cn1 = c6ab_table[i][3];
        const double ref_cn2 = c6ab_table[i][4];

        mxc[idx_atom_1 + 1] = std::max(mxc[idx_atom_1 + 1], grid_i);
        mxc[idx_atom_2 + 1] = std::max(mxc[idx_atom_2 + 1], grid_j);
        c6ab[idx_atom_1 + 1][idx_atom_2 + 1][grid_i - 1][grid_j - 1][0] = ref_c6;
        c6ab[idx_atom_1 + 1][idx_atom_2 + 1][grid_i - 1][grid_j - 1][1] = ref_cn1;
        c6ab[idx_atom_1 + 1][idx_atom_2 + 1][grid_i - 1][grid_j - 1][2] = ref_cn2;
        c6ab[idx_atom_2 + 1][idx_atom_1 + 1][grid_j - 1][grid_i - 1][0] = ref_c6;
        c6ab[idx_atom_2 + 1][idx_atom_1 + 1][grid_j - 1][grid_i - 1][1] = ref_cn2;
        c6ab[idx_atom_2 + 1][idx_atom_1 + 1][grid_j - 1][grid_i - 1][2] = ref_cn1;
    }
}

/* ----------------------------------------------------------------------
   Set functional parameters (used in PairD3::coeff)
------------------------------------------------------------------------- */

void PairD3::setfuncpar_zero() {
    s6 = 1.0;
    alp = 14.0;
    rs18 = 1.0;

    // default def2-QZVP (almost basis set limit)
    std::unordered_map<std::string, int> commandMap = {
    { "slater-dirac-exchange", 1}, { "b-lyp", 2 },    { "b-p", 3 },       { "b97-d", 4 },      { "revpbe", 5 },
    { "pbe", 6 },                  { "pbesol", 7 },   { "rpw86-pbe", 8 }, { "rpbe", 9 },       { "tpss", 10 },
    { "b3-lyp", 11 },              { "pbe0", 12 },    { "hse06", 13 },    { "revpbe38", 14 },  { "pw6b95", 15 },
    { "tpss0", 16 },               { "b2-plyp", 17 }, { "pwpb95", 18 },   { "b2gp-plyp", 19 }, { "ptpss", 20 },
    { "hf", 21 },                  { "mpwlyp", 22 },  { "bpbe", 23 },     { "bh-lyp", 24 },    { "tpssh", 25 },
    { "pwb6k", 26 },               { "b1b95", 27 },   { "bop", 28 },      { "o-lyp", 29 },     { "o-pbe", 30 },
    { "ssb", 31 },                 { "revssb", 32 },  { "otpss", 33 },    { "b3pw91", 34 },    { "revpbe0", 35 },
    { "pbe38", 36 },               { "mpw1b95", 37 }, { "mpwb1k", 38 },   { "bmk", 39 },       { "cam-b3lyp", 40 },
    { "lc-wpbe", 41 },             { "m05", 42 },     { "m052x", 43 },    { "m06l", 44 },      { "m06", 45 },
    { "m062x", 46 },               { "m06hf", 47 },   { "hcth120", 48 }
    };

    int commandCode = commandMap[functional];
    switch (commandCode) {
    case 1: rs6 = 0.999; s18 = -1.957; rs18 = 0.697; break;
    case 2: rs6 = 1.094; s18 = 1.682; break;
    case 3: rs6 = 1.139; s18 = 1.683; break;
    case 4: rs6 = 0.892; s18 = 0.909; break;
    case 5: rs6 = 0.923; s18 = 1.010; break;
    case 6: rs6 = 1.217; s18 = 0.722; break;
    case 7: rs6 = 1.345; s18 = 0.612; break;
    case 8: rs6 = 1.224; s18 = 0.901; break;
    case 9: rs6 = 0.872; s18 = 0.514; break;
    case 10: rs6 = 1.166; s18 = 1.105; break;
    case 11: rs6 = 1.261; s18 = 1.703; break;
    case 12: rs6 = 1.287; s18 = 0.928; break;
    case 13: rs6 = 1.129; s18 = 0.109; break;
    case 14: rs6 = 1.021; s18 = 0.862; break;
    case 15: rs6 = 1.532; s18 = 0.862; break;
    case 16: rs6 = 1.252; s18 = 1.242; break;
    case 17: rs6 = 1.427; s18 = 1.022; s6 = 0.64; break;
    case 18: rs6 = 1.557; s18 = 0.705; s6 = 0.82; break;
    case 19: rs6 = 1.586; s18 = 0.760; s6 = 0.56; break;
    case 20: rs6 = 1.541; s18 = 0.879; s6 = 0.75; break;
    case 21: rs6 = 1.158; s18 = 1.746; break;
    case 22: rs6 = 1.239; s18 = 1.098; break;
    case 23: rs6 = 1.087; s18 = 2.033; break;
    case 24: rs6 = 1.370; s18 = 1.442; break;
    case 25: rs6 = 1.223; s18 = 1.219; break;
    case 26: rs6 = 1.660; s18 = 0.550; break;
    case 27: rs6 = 1.613; s18 = 1.868; break;
    case 28: rs6 = 0.929; s18 = 1.975; break;
    case 29: rs6 = 0.806; s18 = 1.764; break;
    case 30: rs6 = 0.837; s18 = 2.055; break;
    case 31: rs6 = 1.215; s18 = 0.663; break;
    case 32: rs6 = 1.221; s18 = 0.560; break;
    case 33: rs6 = 1.128; s18 = 1.494; break;
    case 34: rs6 = 1.176; s18 = 1.775; break;
    case 35: rs6 = 0.949; s18 = 0.792; break;
    case 36: rs6 = 1.333; s18 = 0.998; break;
    case 37: rs6 = 1.605; s18 = 1.118; break;
    case 38: rs6 = 1.671; s18 = 1.061; break;
    case 39: rs6 = 1.931; s18 = 2.168; break;
    case 40: rs6 = 1.378; s18 = 1.217; break;
    case 41: rs6 = 1.355; s18 = 1.279; break;
    case 42: rs6 = 1.373; s18 = 0.595; break;
    case 43: rs6 = 1.417; s18 = 0.000; break;
    case 44: rs6 = 1.581; s18 = 0.000; break;
    case 45: rs6 = 1.325; s18 = 0.000; break;
    case 46: rs6 = 1.619; s18 = 0.000; break;
    case 47: rs6 = 1.446; s18 = 0.000; break;
    /* DFTB3(zeta = 4.0), old deprecated parameters; case ("dftb3"); rs6 = 1.235; s18 = 0.673; */
    case 48: rs6 = 1.221; s18 = 1.206; break;
    default:
        error->all(FLERR, "Functional name unknown");
        break;
    }
}

void PairD3::setfuncpar_bj() {
    s6 = 1.0;
    alp = 14.0;

    std::unordered_map<std::string, int> commandMap = {
        {"b-p", 1}, {"b-lyp", 2}, {"revpbe", 3}, {"rpbe", 4}, {"b97-d", 5}, {"pbe", 6},
        {"rpw86-pbe", 7}, {"b3-lyp", 8}, {"tpss", 9}, {"hf", 10}, {"tpss0", 11}, {"pbe0", 12},
        {"hse06", 13}, {"revpbe38", 14}, {"pw6b95", 15}, {"b2-plyp", 16}, {"dsd-blyp", 17},
        {"dsd-blyp-fc", 18}, {"bop", 19}, {"mpwlyp", 20}, {"o-lyp", 21}, {"pbesol", 22}, {"bpbe", 23},
        {"opbe", 24}, {"ssb", 25}, {"revssb", 26}, {"otpss", 27}, {"b3pw91", 28}, {"bh-lyp", 29},
        {"revpbe0", 30}, {"tpssh", 31}, {"mpw1b95", 32}, {"pwb6k", 33}, {"b1b95", 34}, {"bmk", 35},
        {"cam-b3lyp", 36}, {"lc-wpbe", 37}, {"b2gp-plyp", 38}, {"ptpss", 39}, {"pwpb95", 40},
        {"hf/mixed", 41}, {"hf/sv", 42}, {"hf/minis", 43}, {"b3-lyp/6-31gd", 44}, {"hcth120", 45},
        {"pw1pw", 46}, {"pwgga", 47}, {"hsesol", 48}, {"hf3c", 49}, {"hf3cv", 50}, {"pbeh3c", 51},
        {"pbeh-3c", 52}, {"wb97m", 53}
    };

    int commandCode = commandMap[functional];
    switch (commandCode) {
        case 1: rs6 = 0.3946; s18 = 3.2822; rs18 = 4.8516; break;
        case 2: rs6 = 0.4298; s18 = 2.6996; rs18 = 4.2359; break;
        case 3: rs6 = 0.5238; s18 = 2.3550; rs18 = 3.5016; break;
        case 4: rs6 = 0.1820; s18 = 0.8318; rs18 = 4.0094; break;
        case 5: rs6 = 0.5545; s18 = 2.2609; rs18 = 3.2297; break;
        case 6: rs6 = 0.4289; s18 = 0.7875; rs18 = 4.4407; break;
        case 7: rs6 = 0.4613; s18 = 1.3845; rs18 = 4.5062; break;
        case 8: rs6 = 0.3981; s18 = 1.9889; rs18 = 4.4211; break;
        case 9: rs6 = 0.4535; s18 = 1.9435; rs18 = 4.4752; break;
        case 10: rs6 = 0.3385; s18 = 0.9171; rs18 = 2.8830; break;
        case 11: rs6 = 0.3768; s18 = 1.2576; rs18 = 4.5865; break;
        case 12: rs6 = 0.4145; s18 = 1.2177; rs18 = 4.8593; break;
        case 13: rs6 = 0.383; s18 = 2.310; rs18 = 5.685; break;
        case 14: rs6 = 0.4309; s18 = 1.4760; rs18 = 3.9446; break;
        case 15: rs6 = 0.2076; s18 = 0.7257; rs18 = 6.3750; break;
        case 16: rs6 = 0.3065; s18 = 0.9147; rs18 = 5.0570; break; s6 = 0.64;
        case 17: rs6 = 0.0000; s18 = 0.2130; rs18 = 6.0519; s6 = 0.50; break;
        case 18: rs6 = 0.0009; s18 = 0.2112; rs18 = 5.9807; s6 = 0.50; break;
        case 19: rs6 = 0.4870; s18 = 3.2950; rs18 = 3.5043; break;
        case 20: rs6 = 0.4831; s18 = 2.0077; rs18 = 4.5323; break;
        case 21: rs6 = 0.5299; s18 = 2.6205; rs18 = 2.8065; break;
        case 22: rs6 = 0.4466; s18 = 2.9491; rs18 = 6.1742; break;
        case 23: rs6 = 0.4567; s18 = 4.0728; rs18 = 4.3908; break;
        case 24: rs6 = 0.5512; s18 = 3.3816; rs18 = 2.9444; break;
        case 25: rs6 = -0.0952; s18 = -0.1744; rs18 = 5.2170; break;
        case 26: rs6 = 0.4720; s18 = 0.4389; rs18 = 4.0986; break;
        case 27: rs6 = 0.4634; s18 = 2.7495; rs18 = 4.3153; break;
        case 28: rs6 = 0.4312; s18 = 2.8524; rs18 = 4.4693; break;
        case 29: rs6 = 0.2793; s18 = 1.0354; rs18 = 4.9615; break;
        case 30: rs6 = 0.4679; s18 = 1.7588; rs18 = 3.7619; break;
        case 31: rs6 = 0.4529; s18 = 2.2382; rs18 = 4.6550; break;
        case 32: rs6 = 0.1955; s18 = 1.0508; rs18 = 6.4177; break;
        case 33: rs6 = 0.1805; s18 = 0.9383; rs18 = 7.7627; break;
        case 34: rs6 = 0.2092; s18 = 1.4507; rs18 = 5.5545; break;
        case 35: rs6 = 0.1940; s18 = 2.0860; rs18 = 5.9197; break;
        case 36: rs6 = 0.3708; s18 = 2.0674; rs18 = 5.4743; break;
        case 37: rs6 = 0.3919; s18 = 1.8541; rs18 = 5.0897; break;
        case 38: rs6 = 0.0000; s18 = 0.2597; rs18 = 6.3332; s6 = 0.560; break;
        case 39: rs6 = 0.0000; s18 = 0.2804; rs18 = 6.5745; s6 = 0.750; break;
        case 40: rs6 = 0.0000; s18 = 0.2904; rs18 = 7.3141; s6 = 0.820; break;
        // special HF / DFT with eBSSE correction;
        case 41: rs6 = 0.5607; s18 = 3.9027; rs18 = 4.5622; break;
        case 42: rs6 = 0.4249; s18 = 2.1849; rs18 = 4.2783; break;
        case 43: rs6 = 0.1702; s18 = 0.9841; rs18 = 3.8506; break;
        case 44: rs6 = 0.5014; s18 = 4.0672; rs18 = 4.8409; break;
        case 45: rs6 = 0.3563; s18 = 1.0821; rs18 = 4.3359; break;
        /*     DFTB3 old, deprecated parameters : ;
            *     case ("dftb3"); rs6 = 0.7461; s18 = 3.209; rs18 = 4.1906;
            *     special SCC - DFTB parametrization;
            *     full third order DFTB, self consistent charges, hydrogen pair damping with; exponent 4.2;
        */
        case 46: rs6 = 0.3807; s18 = 2.3363; rs18 = 5.8844; break;
        case 47: rs6 = 0.2211; s18 = 2.6910; rs18 = 6.7278; break;
        case 48: rs6 = 0.4650; s18 = 2.9215; rs18 = 6.2003; break;
        // special HF - D3 - gCP - SRB / MINIX parametrization;
        case 49: rs6 = 0.4171; s18 = 0.8777; rs18 = 2.9149; break;
        // special HF - D3 - gCP - SRB2 / ECP - 2G parametrization;
        case 50: rs6 = 0.3063; s18 = 0.5022; rs18 = 3.9856; break;
        // special PBEh - D3 - gCP / def2 - mSVP parametrization;
        case 51: rs6 = 0.4860; s18 = 0.0000; rs18 = 4.5000; break;
        case 52: rs6 = 0.4860; s18 = 0.0000; rs18 = 4.5000; break;
        case 53: rs6 = 0.5660; s18 = 0.3908; rs18 = 3.1280; break;
        default:
            error->all(FLERR, "Functional name unknown");
            break;
    }
}

void PairD3::setfuncpar_zerom() {
    s6 = 1.0;
    alp = 14.0;

    std::unordered_map<std::string, int> commandMap = {
        {"b2-plyp", 1}, {"b3-lyp", 2}, {"b97-d", 3}, {"b-lyp", 4},
        {"b-p", 5}, {"pbe", 6}, {"pbe0", 7}, {"lc-wpbe", 8}
    };

    int commandCode = commandMap[functional];
    switch (commandCode) {
        case 1: rs6 = 1.313134; s18 = 0.717543; rs18 = 0.016035; s6 = 0.640000; break;
        case 2: rs6 = 1.338153; s18 = 1.532981; rs18 = 0.013988; break;
        case 3: rs6 = 1.151808; s18 = 1.020078; rs18 = 0.035964; break;
        case 4: rs6 = 1.279637; s18 = 1.841686; rs18 = 0.014370; break;
        case 5: rs6 = 1.233460; s18 = 1.945174; rs18 = 0.000000; break;
        case 6: rs6 = 2.340218; s18 = 0.000000; rs18 = 0.129434; break;
        case 7: rs6 = 2.077949; s18 = 0.000081; rs18 = 0.116755; break;
        case 8: rs6 = 1.366361; s18 = 1.280619; rs18 = 0.003160; break;
        default:
            error->all(FLERR, "Functional name unknown");
            break;
    }
}

void PairD3::setfuncpar_bjm() {
    s6 = 1.0;
    alp = 14.0;

    std::unordered_map<std::string, int> commandMap = {
        {"b2-plyp", 1}, {"b3-lyp", 2}, {"b97-d", 3}, {"b-lyp", 4},
        {"b-p", 5}, {"pbe", 6}, {"pbe0", 7}, {"lc-wpbe", 8}
    };

    int commandCode = commandMap[functional];
    switch (commandCode) {
        case 1: rs6 = 0.486434; s18 = 0.672820; rs18 = 3.656466; s6 = 0.640000; break;
        case 2: rs6 = 0.278672; s18 = 1.466677; rs18 = 4.606311; break;
        case 3: rs6 = 0.240184; s18 = 1.206988; rs18 = 3.864426; break;
        case 4: rs6 = 0.448486; s18 = 1.875007; rs18 = 3.610679; break;
        case 5: rs6 = 0.821850; s18 = 3.140281; rs18 = 2.728151; break;
        case 6: rs6 = 0.012092; s18 = 0.358940; rs18 = 5.938951; break;
        case 7: rs6 = 0.007912; s18 = 0.528823; rs18 = 6.162326; break;
        case 8: rs6 = 0.563761; s18 = 0.906564; rs18 = 3.593680; break;
        default:
            error->all(FLERR, "Functional name unknown");
            break;
    }
}

void PairD3::setfuncpar() {
    void (PairD3::*setfuncpar_damp[4])() = {
        &PairD3::setfuncpar_zero,
        &PairD3::setfuncpar_bj,
        &PairD3::setfuncpar_zerom,
        &PairD3::setfuncpar_bjm
    };
    (this->*setfuncpar_damp[damping])();

    rs8 = rs18;
    alp6 = alp;
    alp8 = alp + 2.0;
    // rs10 = rs18
    // alp10 = alp + 4.0;

    a1 = rs6;
    a2 = rs8;
    s8 = s18;
    // s6 is already defined
}

/* ----------------------------------------------------------------------
   Coeff : read from pair_coeff (Required) -> pair_coeff * * element1 element2 ...
------------------------------------------------------------------------- */

void PairD3::coeff(int narg, char **arg) {
    if (!allocated) allocate();

    int ntypes = atom->ntypes;
    if (narg != ntypes + 2) { error->all(FLERR, "Pair_coeff needs: * * element1 element2 ..."); }

    std::string element;
    int* atomic_numbers = (int*)malloc(sizeof(int)*ntypes);
    for (int i = 0; i < ntypes; i++) {
        element = arg[i+2];
        atomic_numbers[i] = find_atomic_number(element);
    }

    int count = 0;
    for (int i = 1; i <= ntypes; i++) {
        for (int j = 1; j <= ntypes; j++) {
            setflag[i][j] = 1;
            count++;
        }
    }

    if (count == 0) error->all(FLERR, "Incorrect args for pair coefficients");

    /*
    scale r4/r2 values of the atoms by sqrt(Z)
    sqrt is also globally close to optimum
    together with the factor 1/2 this yield reasonable
    c8 for he, ne and ar. for larger Z, C8 becomes too large
    which effectively mimics higher R^n terms neglected due
    to stability reasons

    r2r4 =sqrt(0.5*r2r4(i)*dfloat(i)**0.5 ) with i=elementnumber
    the large number of digits is just to keep the results consistent
    with older versions. They should not imply any higher accuracy than
    the old values
    */
    double r2r4_ref[94] = {
         2.00734898,  1.56637132,  5.01986934,  3.85379032,  3.64446594,
         3.10492822,  2.71175247,  2.59361680,  2.38825250,  2.21522516,
         6.58585536,  5.46295967,  5.65216669,  4.88284902,  4.29727576,
         4.04108902,  3.72932356,  3.44677275,  7.97762753,  7.07623947,
         6.60844053,  6.28791364,  6.07728703,  5.54643096,  5.80491167,
         5.58415602,  5.41374528,  5.28497229,  5.22592821,  5.09817141,
         6.12149689,  5.54083734,  5.06696878,  4.87005108,  4.59089647,
         4.31176304,  9.55461698,  8.67396077,  7.97210197,  7.43439917,
         6.58711862,  6.19536215,  6.01517290,  5.81623410,  5.65710424,
         5.52640661,  5.44263305,  5.58285373,  7.02081898,  6.46815523,
         5.98089120,  5.81686657,  5.53321815,  5.25477007, 11.02204549,
        10.15679528,  9.35167836,  9.06926079,  8.97241155,  8.90092807,
         8.85984840,  8.81736827,  8.79317710,  7.89969626,  8.80588454,
         8.42439218,  8.54289262,  8.47583370,  8.45090888,  8.47339339,
         7.83525634,  8.20702843,  7.70559063,  7.32755997,  7.03887381,
         6.68978720,  6.05450052,  5.88752022,  5.70661499,  5.78450695,
         7.79780729,  7.26443867,  6.78151984,  6.67883169,  6.39024318,
         6.09527958, 11.79156076, 11.10997644,  9.51377795,  8.67197068,
         8.77140725,  8.65402716,  8.53923501,  8.85024712
    }; // atomic <r^2>/<r^4> values

    /*
    covalent radii (taken from Pyykko and Atsumi, Chem. Eur. J. 15, 2009, 188-197)
    values for metals decreased by 10 %
    !      data rcov/
    !     .  0.32, 0.46, 1.20, 0.94, 0.77, 0.75, 0.71, 0.63, 0.64, 0.67
    !     ., 1.40, 1.25, 1.13, 1.04, 1.10, 1.02, 0.99, 0.96, 1.76, 1.54
    !     ., 1.33, 1.22, 1.21, 1.10, 1.07, 1.04, 1.00, 0.99, 1.01, 1.09
    !     ., 1.12, 1.09, 1.15, 1.10, 1.14, 1.17, 1.89, 1.67, 1.47, 1.39
    !     ., 1.32, 1.24, 1.15, 1.13, 1.13, 1.08, 1.15, 1.23, 1.28, 1.26
    !     ., 1.26, 1.23, 1.32, 1.31, 2.09, 1.76, 1.62, 1.47, 1.58, 1.57
    !     ., 1.56, 1.55, 1.51, 1.52, 1.51, 1.50, 1.49, 1.49, 1.48, 1.53
    !     ., 1.46, 1.37, 1.31, 1.23, 1.18, 1.16, 1.11, 1.12, 1.13, 1.32
    !     ., 1.30, 1.30, 1.36, 1.31, 1.38, 1.42, 2.01, 1.81, 1.67, 1.58
    !     ., 1.52, 1.53, 1.54, 1.55 /

    these new data are scaled with k2=4./3.  and converted a_0 via
    autoang=0.52917726d0
    */

    double rcov_ref[94] = {
        0.80628308, 1.15903197, 3.02356173, 2.36845659, 1.94011865,
        1.88972601, 1.78894056, 1.58736983, 1.61256616, 1.68815527,
        3.52748848, 3.14954334, 2.84718717, 2.62041997, 2.77159820,
        2.57002732, 2.49443835, 2.41884923, 4.43455700, 3.88023730,
        3.35111422, 3.07395437, 3.04875805, 2.77159820, 2.69600923,
        2.62041997, 2.51963467, 2.49443835, 2.54483100, 2.74640188,
        2.82199085, 2.74640188, 2.89757982, 2.77159820, 2.87238349,
        2.94797246, 4.76210950, 4.20778980, 3.70386304, 3.50229216,
        3.32591790, 3.12434702, 2.89757982, 2.84718717, 2.84718717,
        2.72120556, 2.89757982, 3.09915070, 3.22513231, 3.17473967,
        3.17473967, 3.09915070, 3.32591790, 3.30072128, 5.26603625,
        4.43455700, 4.08180818, 3.70386304, 3.98102289, 3.95582657,
        3.93062995, 3.90543362, 3.80464833, 3.82984466, 3.80464833,
        3.77945201, 3.75425569, 3.75425569, 3.72905937, 3.85504098,
        3.67866672, 3.45189952, 3.30072128, 3.09915070, 2.97316878,
        2.92277614, 2.79679452, 2.82199085, 2.84718717, 3.32591790,
        3.27552496, 3.27552496, 3.42670319, 3.30072128, 3.47709584,
        3.57788113, 5.06446567, 4.56053862, 4.20778980, 3.98102289,
        3.82984466, 3.85504098, 3.88023730, 3.90543362
    }; // covalent radii

    for (int i = 0; i < ntypes; i++) {
        r2r4[i+1] = r2r4_ref[atomic_numbers[i]-1];
        rcov[i+1] = rcov_ref[atomic_numbers[i]-1];
    }

    // set r0ab
    read_r0ab(atomic_numbers, ntypes);

    // read c6ab
    read_c6ab(atomic_numbers, ntypes);

    free(atomic_numbers);
}

/* ----------------------------------------------------------------------
   Get derivative of C6 w.r.t. CN (used in PairD3::compute)

   C6 = C6(CN_A, CN_B) == W(CN_A, CN_B) / Z(CN_A, CN_B)

   This gives below from chain rule:
   d(C6)/dr = d(C6)/d(CN_A) * d(CN_A)/dr + d(C6)/d(CN_B) * d(CN_B)/dr

   So we can pre-calculate the d(C6)/d(CN_A), d(C6)/d(CN_B) part.

   d(C6)/d(CN_i) = (dW/d(CN_i) * Z - W * dZ/d(CN_i)) / (W * W)
        W : "denominator"
        Z : "numerator"
        dW/d(CN_i) : "d_denominator_i"
        dZ/d(CN_j) : "d_numerator_j"

    Z = Sum( L_ij(CN_A, CN_B) * C6_ref(CN_A_i, CN_B_j) ) over i, j
    W = Sum( L_ij(CN_A, CN_B) ) over i, j

   And the resulting derivative term is saved into
   "dc6_iji_tot", "dc6_ijj_tot" array,
   where we can find the value of d(C6)/d(CN_i)
   by knowing the index of "iat", and "jat". ("idx_linij")

   Also, c6 values will also be saved into "c6_ij_tot" array.

   Here, as we only interested in *pair* of atoms, assume "iat" >= "jat".
   Then "idx_linij" = "jat + (iat + 1) * iat / 2" have the order below.

     idx_linij | j = 0  j = 1  j = 2  j = 3    ...
---------------------------------------------
        i = 0  |     0
        i = 1  |     1      2
        i = 2  |     3      4      5
        i = 3  |     6      7      8      9
          ...  |    ...    ...    ...    ...   ...

------------------------------------------------------------------------- */

__global__ void kernel_get_dC6_dCNij(
    int maxij, float K3,
    double *cn, int *mxc, float *****c6ab, int *type,
    float *c6_ij_tot, float *dc6_iji_tot, float *dc6_ijj_tot
) {
    int iter = blockIdx.x * blockDim.x + threadIdx.x;

    if (iter < maxij) {
        int iat, jat;
        ij_at_linij(iter, iat, jat);

        const int atomtype_i = type[iat];
        const int atomtype_j = type[jat];

        const float cni = cn[iat];
        const int mxci = mxc[atomtype_i];
        const float cnj = cn[jat];
        const int mxcj = mxc[atomtype_j];

        float c6mem = -1e99f;
        float r_save = 9999.0f;
        double numerator = 0.0;
        double denominator = 0.0;
        double d_numerator_i = 0.0;
        double d_denominator_i = 0.0;
        double d_numerator_j = 0.0;
        double d_denominator_j = 0.0;

        for (int a = 0; a < mxci; a++) {
            for (int b = 0; b < mxcj; b++) {
                float c6ref = c6ab[atomtype_i][atomtype_j][a][b][0];

                if (c6ref > 0.0f) {
                    float cn_refi = c6ab[atomtype_i][atomtype_j][a][b][1];
                    float cn_refj = c6ab[atomtype_i][atomtype_j][a][b][2];

                    float r = (cn_refi - cni) * (cn_refi - cni) + (cn_refj - cnj) * (cn_refj - cnj);
                    if (r < r_save) {
                        r_save = r;
                        c6mem = c6ref;
                    }

                    double expterm = exp(static_cast<double>(K3) * static_cast<double>(r)); // must be double
                    numerator += c6ref * expterm;
                    denominator += expterm;

                    expterm *= 2.0f * K3;

                    double term = expterm * (cni - cn_refi);
                    d_numerator_i += c6ref * term;
                    d_denominator_i += term;

                    term = expterm * (cnj - cn_refj);
                    d_numerator_j += c6ref * term;
                    d_denominator_j += term;
                }
            }
        }

        if (denominator > 1e-99) {
            const double denominator_rc = 1.0 / denominator; // must be double
            const double unit_frac = numerator * denominator_rc;
            c6_ij_tot[iter] = unit_frac;
            dc6_iji_tot[iter] = denominator_rc * fma(unit_frac, -d_denominator_i, d_numerator_i); // must be double
            dc6_ijj_tot[iter] = denominator_rc * fma(unit_frac, -d_denominator_j, d_numerator_j); // must be double
            //const double denominator_rc = 1.0 / denominator;
            //const float unit_frac = numerator * denominator_rc;
            //c6_ij_tot[iter] = unit_frac;
            //dc6_iji_tot[iter] = \
            static_cast<float>(d_numerator_i * denominator_rc) - static_cast<float>(d_denominator_i * denominator_rc) * unit_frac;
            //dc6_ijj_tot[iter] = \
            static_cast<float>(d_numerator_j * denominator_rc) - static_cast<float>(d_denominator_j * denominator_rc) * unit_frac;
        }
        else {
            c6_ij_tot[iter] = c6mem;
            dc6_iji_tot[iter] = 0.0f;
            dc6_ijj_tot[iter] = 0.0f;
        }
    }
}

void PairD3::get_dC6_dCNij() {
    int n = atom->natoms;
    int maxij = n * (n + 1) / 2;

    //START_CUDA_TIMER();

    int threadsPerBlock = 128;
    int blocksPerGrid = (maxij + threadsPerBlock - 1) / threadsPerBlock;
    kernel_get_dC6_dCNij<<<blocksPerGrid, threadsPerBlock>>>(
        maxij, K3,
        cn, mxc, c6ab, atomtype,
        c6_ij_tot, dc6_iji_tot, dc6_ijj_tot
    );
    hipDeviceSynchronize();

    //STOP_CUDA_TIMER("get_dC6dCNij");
}

/* ----------------------------------------------------------------------
   Get lattice vectors (used in PairD3::compute)

   1) Save lattice vectors into "lat_v_1", "lat_v_2", "lat_v_3"
   2) Calculate repetition criteria for vdw, cn
   3) precaluclate tau (xyz shift due to cell repetition)

------------------------------------------------------------------------- */

void PairD3::set_lattice_vectors() {
    double boxxlo = domain->boxlo[0];
    double boxxhi = domain->boxhi[0];
    double boxylo = domain->boxlo[1];
    double boxyhi = domain->boxhi[1];
    double boxzlo = domain->boxlo[2];
    double boxzhi = domain->boxhi[2];
    double xy = domain->xy;
    double xz = domain->xz;
    double yz = domain->yz;

    lat_v_1[0] = (boxxhi - boxxlo) / AU_TO_ANG;
    lat_v_1[1] =               0.0;
    lat_v_1[2] =               0.0;
    lat_v_2[0] =                xy / AU_TO_ANG;
    lat_v_2[1] = (boxyhi - boxylo) / AU_TO_ANG;
    lat_v_2[2] =               0.0;
    lat_v_3[0] =                xz / AU_TO_ANG;
    lat_v_3[1] =                yz / AU_TO_ANG;
    lat_v_3[2] = (boxzhi - boxzlo) / AU_TO_ANG;

    int vdwrx_save = 2 * rep_vdw[0] + 1;
    int vdwry_save = 2 * rep_vdw[1] + 1;
    int vdwrz_save = 2 * rep_vdw[2] + 1;
    int cnrx_save = 2 * rep_cn[0] + 1;
    int cnry_save = 2 * rep_cn[1] + 1;
    int cnrz_save = 2 * rep_cn[2] + 1;

    set_lattice_repetition_criteria(rthr, rep_vdw);
    set_lattice_repetition_criteria(cnthr, rep_cn);

    int vdw_range_x = 2 * rep_vdw[0] + 1;
    int vdw_range_y = 2 * rep_vdw[1] + 1;
    int vdw_range_z = 2 * rep_vdw[2] + 1;
    int tau_loop_size_vdw = vdw_range_x * vdw_range_y * vdw_range_z * 3;
    if (tau_loop_size_vdw != tau_idx_vdw_total_size) {
        if (tau_idx_vdw != nullptr) {
            for (int i = 0; i < vdwrx_save; i++) {
                for (int j = 0; j < vdwry_save; j++) {
                    for (int k = 0; k < vdwrz_save; k++) {
                        hipFree(tau_vdw[i][j][k]);
                    }
                    hipFree(tau_vdw[i][j]);
                }
                hipFree(tau_vdw[i]);
            }
            hipFree(tau_vdw);
            hipFree(tau_idx_vdw);
        }
        tau_idx_vdw_total_size = tau_loop_size_vdw;
        hipMallocManaged(&tau_vdw, vdw_range_x * sizeof(float***));
        for (int i = 0; i < vdw_range_x; i++) {
            hipMallocManaged(&tau_vdw[i], vdw_range_y * sizeof(float**));
            for (int j = 0; j < vdw_range_y; j++) {
                hipMallocManaged(&tau_vdw[i][j], vdw_range_z * sizeof(float*));
                for (int k = 0; k < vdw_range_z; k++) {
                    hipMallocManaged(&tau_vdw[i][j][k], 3 * sizeof(float));
                }
            }
        }
        hipMallocManaged(&tau_idx_vdw, tau_idx_vdw_total_size * sizeof(int));
    }

    int cn_range_x  = 2 * rep_cn[0] + 1;
    int cn_range_y  = 2 * rep_cn[1] + 1;
    int cn_range_z  = 2 * rep_cn[2] + 1;
    int tau_loop_size_cn = cn_range_x * cn_range_y * cn_range_z * 3;
    if (tau_loop_size_cn != tau_idx_cn_total_size) {
        if (tau_idx_cn != nullptr) {
            for (int i = 0; i < cnrx_save; i++) {
                for (int j = 0; j < cnry_save; j++) {
                    for (int k = 0; k < cnrz_save; k++) {
                        hipFree(tau_cn[i][j][k]);
                    }
                    hipFree(tau_cn[i][j]);
                }
                hipFree(tau_cn[i]);
            }
            hipFree(tau_cn);
            hipFree(tau_idx_cn);
        }
        tau_idx_cn_total_size = tau_loop_size_cn;
        hipMallocManaged(&tau_cn, cn_range_x * sizeof(float***));
        for (int i = 0; i < cn_range_x; i++) {
            hipMallocManaged(&tau_cn[i], cn_range_y * sizeof(float**));
            for (int j = 0; j < cn_range_y; j++) {
                hipMallocManaged(&tau_cn[i][j], cn_range_z * sizeof(float*));
                for (int k = 0; k < cn_range_z; k++) {
                    hipMallocManaged(&tau_cn[i][j][k], 3 * sizeof(float));
                }
            }
        }
        hipMallocManaged(&tau_idx_cn, tau_idx_cn_total_size * sizeof(int));
    }
}

/* ----------------------------------------------------------------------
   Set repetition criteria (used in PairD3::compute)

   Needed as Periodic Boundary Condition should be considered.

   As the cell may *not* be orthorhombic,
   the dot product should be used between x/y/z direction and
   corresponding cross product vector.
------------------------------------------------------------------------- */

void PairD3::set_lattice_repetition_criteria(float r_threshold, int* rep_v) {
    double r_cutoff = sqrt(r_threshold);
    double lat_cp_12[3], lat_cp_23[3], lat_cp_31[3];
    double cos_value;

    MathExtra::cross3(lat_v_1, lat_v_2, lat_cp_12);
    MathExtra::cross3(lat_v_2, lat_v_3, lat_cp_23);
    MathExtra::cross3(lat_v_3, lat_v_1, lat_cp_31);

    cos_value = MathExtra::dot3(lat_cp_23, lat_v_1) / MathExtra::len3(lat_cp_23);
    rep_v[0] = static_cast<int>(std::abs(r_cutoff / cos_value)) + 1;
    cos_value = MathExtra::dot3(lat_cp_31, lat_v_2) / MathExtra::len3(lat_cp_31);
    rep_v[1] = static_cast<int>(std::abs(r_cutoff / cos_value)) + 1;
    cos_value = MathExtra::dot3(lat_cp_12, lat_v_3) / MathExtra::len3(lat_cp_12);
    rep_v[2] = static_cast<int>(std::abs(r_cutoff / cos_value)) + 1;

    if (domain->xperiodic == 0) { rep_v[0] = 0; }
    if (domain->yperiodic == 0) { rep_v[1] = 0; }
    if (domain->zperiodic == 0) { rep_v[2] = 0; }
}

/* ----------------------------------------------------------------------
   Calculate Coordination Number (used in PairD3::compute)
------------------------------------------------------------------------- */

__global__ void kernel_get_coordination_number(
    int maxij, int maxtau, float cnthr, float K1,
    float *rcov, int *rep_cn, float ****tau_cn, int *tau_idx_cn, int *type, float **x,
    double *cn
) {
    int iter = blockIdx.x * blockDim.x + threadIdx.x;

    if (iter < maxij) {
        int iat, jat;
        ij_at_linij(iter, iat, jat);

        float cn_local = 0.0f;

        if (iat == jat) {
            const float rcov_sum = rcov[type[iat]] * 2.0f;
            for (int k = maxtau - 1; k >= 0; k -= 3) {
                const int idx1 = tau_idx_cn[k-2];
                const int idx2 = tau_idx_cn[k-1];
                const int idx3 = tau_idx_cn[k];
                if (idx1 == rep_cn[0] && idx2 == rep_cn[1] && idx3 == rep_cn[2]) { continue; }
                const float rx = tau_cn[idx1][idx2][idx3][0];
                const float ry = tau_cn[idx1][idx2][idx3][1];
                const float rz = tau_cn[idx1][idx2][idx3][2];
                const float r2 = rx * rx + ry * ry + rz * rz;
                if (r2 <= cnthr) {
                    const float r_rc = rsqrtf(r2);
                    const float damp = 1.0f / (1.0f + expf(-K1 * ((rcov_sum * r_rc) - 1.0f)));
                    cn_local += damp;
                }
            }
            atomicAdd(&cn[iat], cn_local);
        }

        else {
            const float rcov_sum = rcov[type[iat]] + rcov[type[jat]];
            for (int k = maxtau - 1; k >= 0; k -= 3) {
                const int idx1 = tau_idx_cn[k-2];
                const int idx2 = tau_idx_cn[k-1];
                const int idx3 = tau_idx_cn[k];
                const float rx = x[jat][0] - x[iat][0] + tau_cn[idx1][idx2][idx3][0];
                const float ry = x[jat][1] - x[iat][1] + tau_cn[idx1][idx2][idx3][1];
                const float rz = x[jat][2] - x[iat][2] + tau_cn[idx1][idx2][idx3][2];
                const float r2 = rx * rx + ry * ry + rz * rz;
                if (r2 <= cnthr) {
                    const float r_rc = rsqrtf(r2);
                    const float damp = 1.0f / (1.0f + expf(-K1 * ((rcov_sum * r_rc) - 1.0f)));
                    cn_local += damp;
                }
            }
            atomicAdd(&cn[iat], cn_local);
            atomicAdd(&cn[jat], cn_local);
        }
    }
}

void PairD3::get_coordination_number() {
    int n = atom->natoms;
    int maxij = n * (n + 1) / 2;
    int maxtau = tau_idx_cn_total_size;

    for (int i = 0; i < n; i++) {
        cn[i] = 0.0;
    }

    //START_CUDA_TIMER();

    int threadsPerBlock = 128;
    int blocksPerGrid = (maxij + threadsPerBlock - 1) / threadsPerBlock;
    kernel_get_coordination_number<<<blocksPerGrid, threadsPerBlock>>>(
        maxij, maxtau, cnthr, K1,
        rcov, rep_cn, tau_cn, tau_idx_cn, atomtype, x,
        cn
    );
    hipDeviceSynchronize();

    //STOP_CUDA_TIMER("get_coord");
}

/* ----------------------------------------------------------------------
   reallcate memory if the number of atoms has changed (used in PairD3::compute)
------------------------------------------------------------------------- */

void PairD3::reallocate_arrays() {
    /* -------------- Destroy previous arrays -------------- */
    hipFree(cn);
    for (int i = 0; i < n_save; i++) { hipFree(x[i]); }; hipFree(x);
    hipFree(dc6i);
    for (int i = 0; i < n_save; i++) { hipFree(f[i]); }; hipFree(f);

    hipFree(dc6_iji_tot);
    hipFree(dc6_ijj_tot);
    hipFree(c6_ij_tot);

    hipFree(atomtype);
    /* -------------- Destroy previous arrays -------------- */

    /* -------------- Create new arrays -------------- */
    int n = atom->natoms;
    n_save = n;

    hipMallocManaged(&cn, n * sizeof(double));
    hipMallocManaged(&x, n * sizeof(float*)); for (int i = 0; i < n; i++) { hipMallocManaged(&x[i], 3 * sizeof(float)); }
    hipMallocManaged(&dc6i, n * sizeof(double));
    hipMallocManaged(&f, n * sizeof(double*)); for (int i = 0; i < n; i++) { hipMallocManaged(&f[i], 3 * sizeof(double)); }

    int n_ij_combination = n * (n + 1) / 2;
    hipMallocManaged(&dc6_iji_tot, n_ij_combination * sizeof(float));
    hipMallocManaged(&dc6_ijj_tot, n_ij_combination * sizeof(float));
    hipMallocManaged(&c6_ij_tot,   n_ij_combination * sizeof(float));

    hipMallocManaged(&atomtype, n * sizeof(int));
    /* -------------- Create new arrays -------------- */
}

/* ----------------------------------------------------------------------
  Initialize atomic positions & types (used in PairD3::compute)

   As the default xyz from lammps does not assure that atoms are within unit cell,
   this function shifts atoms into the unit cell.
------------------------------------------------------------------------- */

void PairD3::load_atom_info() {
    double lat[3][3];
    lat[0][0] = lat_v_1[0];
    lat[0][1] = lat_v_2[0];
    lat[0][2] = lat_v_3[0];
    lat[1][0] = lat_v_1[1];
    lat[1][1] = lat_v_2[1];
    lat[1][2] = lat_v_3[1];
    lat[2][0] = lat_v_1[2];
    lat[2][1] = lat_v_2[2];
    lat[2][2] = lat_v_3[2];

    double det = lat[0][0] * lat[1][1] * lat[2][2]
               + lat[0][1] * lat[1][2] * lat[2][0]
               + lat[0][2] * lat[1][0] * lat[2][1]
               - lat[0][2] * lat[1][1] * lat[2][0]
               - lat[0][1] * lat[1][0] * lat[2][2]
               - lat[0][0] * lat[1][2] * lat[2][1];

    double lat_inv[3][3];
    lat_inv[0][0] = (lat[1][1] * lat[2][2] - lat[1][2] * lat[2][1]) / det;
    lat_inv[1][0] = (lat[1][2] * lat[2][0] - lat[1][0] * lat[2][2]) / det;
    lat_inv[2][0] = (lat[1][0] * lat[2][1] - lat[1][1] * lat[2][0]) / det;
    lat_inv[0][1] = (lat[0][2] * lat[2][1] - lat[0][1] * lat[2][2]) / det;
    lat_inv[1][1] = (lat[0][0] * lat[2][2] - lat[0][2] * lat[2][0]) / det;
    lat_inv[2][1] = (lat[0][1] * lat[2][0] - lat[0][0] * lat[2][1]) / det;
    lat_inv[0][2] = (lat[0][1] * lat[1][2] - lat[0][2] * lat[1][1]) / det;
    lat_inv[1][2] = (lat[0][2] * lat[1][0] - lat[0][0] * lat[1][2]) / det;
    lat_inv[2][2] = (lat[0][0] * lat[1][1] - lat[0][1] * lat[1][0]) / det;

    double a[3] = { 0.0 };
    for (int iat = 0; iat < atom->natoms; iat++) {
        for (int i = 0; i < 3; i++) {
            a[i] = lat_inv[i][0] * (atom->x)[iat][0] / AU_TO_ANG +
                   lat_inv[i][1] * (atom->x)[iat][1] / AU_TO_ANG +
                   lat_inv[i][2] * (atom->x)[iat][2] / AU_TO_ANG;
            a[i] -= floor(a[i]); // replaces the code below
            //if      (a[i] > 1) { while (a[i] > 1) { a[i]--; } }
            //else if (a[i] < 0) { while (a[i] < 0) { a[i]++; } }
        }

        for (int i = 0; i < 3; i++) {
            x[iat][i] = (lat[i][0] * a[0] + lat[i][1] * a[1] + lat[i][2] * a[2]);
        }
    }
}

/* ----------------------------------------------------------------------
   Precalculate tau array
------------------------------------------------------------------------- */

void PairD3::precalculate_tau_array() {
    int xlim = rep_vdw[0];
    int ylim = rep_vdw[1];
    int zlim = rep_vdw[2];

    int index = 0;
    for (int taux = -xlim; taux <= xlim; taux++) {
        for (int tauy = -ylim; tauy <= ylim; tauy++) {
            for (int tauz = -zlim; tauz <= zlim; tauz++) {
                tau_vdw[taux + xlim][tauy + ylim][tauz + zlim][0] = lat_v_1[0] * taux + lat_v_2[0] * tauy + lat_v_3[0] * tauz;
                tau_vdw[taux + xlim][tauy + ylim][tauz + zlim][1] = lat_v_1[1] * taux + lat_v_2[1] * tauy + lat_v_3[1] * tauz;
                tau_vdw[taux + xlim][tauy + ylim][tauz + zlim][2] = lat_v_1[2] * taux + lat_v_2[2] * tauy + lat_v_3[2] * tauz;
                tau_idx_vdw[index++] = taux + xlim;
                tau_idx_vdw[index++] = tauy + ylim;
                tau_idx_vdw[index++] = tauz + zlim;
            }
        }
    }

    xlim = rep_cn[0];
    ylim = rep_cn[1];
    zlim = rep_cn[2];

    index = 0;
    for (int taux = -xlim; taux <= xlim; taux++) {
        for (int tauy = -ylim; tauy <= ylim; tauy++) {
            for (int tauz = -zlim; tauz <= zlim; tauz++) {
                tau_cn[taux + xlim][tauy + ylim][tauz + zlim][0] = lat_v_1[0] * taux + lat_v_2[0] * tauy + lat_v_3[0] * tauz;
                tau_cn[taux + xlim][tauy + ylim][tauz + zlim][1] = lat_v_1[1] * taux + lat_v_2[1] * tauy + lat_v_3[1] * tauz;
                tau_cn[taux + xlim][tauy + ylim][tauz + zlim][2] = lat_v_1[2] * taux + lat_v_2[2] * tauy + lat_v_3[2] * tauz;
                tau_idx_cn[index++] = taux + xlim;
                tau_idx_cn[index++] = tauy + ylim;
                tau_idx_cn[index++] = tauz + zlim;
            }
        }
    }
}

/* ----------------------------------------------------------------------
   Get forces (Zero damping)
------------------------------------------------------------------------- */

__global__ void kernel_get_forces_without_dC6_zero(
    int maxij, int maxtau, float rthr, float s6, float s8, float a1, float a2, float alp6, float alp8,
    float *r2r4, float **r0ab, int *rep_vdw, float ****tau_vdw, int *tau_idx_vdw, int *type, float **x,
    float *c6_ij_tot, float *dc6_iji_tot, float *dc6_ijj_tot,
    double *dc6i, double *disp, double **f, double **sigma
) {
    int iter = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ float sigma_00[128];
    __shared__ float sigma_01[128];
    __shared__ float sigma_02[128];
    __shared__ float sigma_10[128];
    __shared__ float sigma_11[128];
    __shared__ float sigma_12[128];
    __shared__ float sigma_20[128];
    __shared__ float sigma_21[128];
    __shared__ float sigma_22[128];
    __shared__ float disp_shared[128];

    float sigma_local_00 = 0.0f;
    float sigma_local_01 = 0.0f;
    float sigma_local_02 = 0.0f;
    float sigma_local_10 = 0.0f;
    float sigma_local_11 = 0.0f;
    float sigma_local_12 = 0.0f;
    float sigma_local_20 = 0.0f;
    float sigma_local_21 = 0.0f;
    float sigma_local_22 = 0.0f;
    float disp_local = 0.0f;

    if (iter < maxij) {
        int iat, jat;
        ij_at_linij(iter, iat, jat);

        float f_local[3] = { 0.0f };
        float dc6i_local_i = 0.0f;
        float dc6i_local_j = 0.0f;

        const float c6 = c6_ij_tot[iter];
        const float dc6iji = dc6_iji_tot[iter];
        const float dc6ijj = dc6_ijj_tot[iter];

        if (iat == jat) {
            const int atomtype_i = type[iat];
            const float r0 = r0ab[atomtype_i][atomtype_i];
            const float unit_r2r4 = r2r4[atomtype_i];
            const float r42 = unit_r2r4 * unit_r2r4;
            const float unit_a1 = (a1 * r0);
            const float unit_a2 = (a2 * r0);
            const float s8r42 = s8 * r42;

            for (int k = maxtau - 1; k >= 0; k -= 3) {
                const int idx1 = tau_idx_vdw[k-2];
                const int idx2 = tau_idx_vdw[k-1];
                const int idx3 = tau_idx_vdw[k];

                if (idx1 == rep_vdw[0] && idx2 == rep_vdw[1] && idx3 == rep_vdw[2]) { continue; }
                const float rij[3] = {
                    tau_vdw[idx1][idx2][idx3][0],
                    tau_vdw[idx1][idx2][idx3][1],
                    tau_vdw[idx1][idx2][idx3][2]
                };
                const float r2 = lensq3(rij);
                if (r2 > rthr) { continue; }

                const float r_rc = rsqrtf(r2);
                float unit_rc_a1 = unit_a1 * r_rc;
                float t6 = unit_rc_a1 * unit_rc_a1; // ^2
                t6 *= unit_rc_a1; // ^3
                t6 *= t6; // ^6
                t6 *= unit_rc_a1; // ^7
                t6 *= t6; // ^14
                const float damp6 = 1.0f / fmaf(t6, 6.0f, 1.0f);
                float unit_rc_a2 = unit_a2 * r_rc;
                float t8 = unit_rc_a2 * unit_rc_a2; // ^2
                t8 *= t8; // ^4
                t8 *= t8; // ^8
                t8 *= t8; // ^16
                const float damp8 = 1.0f / fmaf(t8, 6.0f, 1.0f);
                const float r2_rc = r_rc * r_rc; // 1.0 / r2
                const float r6_rc = r2_rc * r2_rc * r2_rc;
                const float r8_rc = r6_rc * r2_rc;
                const float x1 = 3.0f * c6 * r8_rc * fmaf(r2_rc, s8r42 * damp8 * fmaf(3.0f * alp8 * t8, damp8, -4.0f), s6 * damp6 * fmaf(alp6 * t6, damp6, -1.0f));
                //const float x1 = 0.5 * 6.0 * c6 * r8_rc * (s6 * damp6 * (14.0 * t6 * damp6 - 1.0) + s8r42 * r2_rc * damp8 * (48.0 * t8 * damp8 - 4.0));
                //3.0 * alp6 = 48.0

                const float vec[3] = {
                    x1 * rij[0],
                    x1 * rij[1],
                    x1 * rij[2]
                };

                sigma_local_00 += vec[0] * rij[0];
                sigma_local_01 += vec[0] * rij[1];
                sigma_local_02 += vec[0] * rij[2];
                sigma_local_10 += vec[1] * rij[0];
                sigma_local_11 += vec[1] * rij[1];
                sigma_local_12 += vec[1] * rij[2];
                sigma_local_20 += vec[2] * rij[0];
                sigma_local_21 += vec[2] * rij[1];
                sigma_local_22 += vec[2] * rij[2];

                const float dc6_rest = 0.5f * r6_rc * fmaf(3.0f * r2_rc, s8r42 * damp8, s6 * damp6);
                //const float dc6_rest = 0.5 * r6_rc * (s6 * damp6 + 3.0 * s8r42 * damp8 * r2_rc);
                disp_local -= dc6_rest * c6;
                dc6i_local_i += dc6_rest * dc6iji;
                dc6i_local_j += dc6_rest * dc6ijj;
            }
            atomicAdd(&dc6i[iat], dc6i_local_i);
            atomicAdd(&dc6i[jat], dc6i_local_j);
        }

        else {
            const int atomtype_i = type[iat];
            const int atomtype_j = type[jat];
            const float r0 = r0ab[atomtype_i][atomtype_j];
            const float r42 = r2r4[atomtype_i] * r2r4[atomtype_j];
            const float unit_a1 = (a1 * r0);
            const float unit_a2 = (a2 * r0);
            const float s8r42 = s8 * r42;

            for (int k = maxtau - 1; k >= 0; k -= 3) {
                const int idx1 = tau_idx_vdw[k-2];
                const int idx2 = tau_idx_vdw[k-1];
                const int idx3 = tau_idx_vdw[k];

                const float rij[3] = {
                    x[jat][0] - x[iat][0] + tau_vdw[idx1][idx2][idx3][0],
                    x[jat][1] - x[iat][1] + tau_vdw[idx1][idx2][idx3][1],
                    x[jat][2] - x[iat][2] + tau_vdw[idx1][idx2][idx3][2]
                };
                const float r2 = lensq3(rij);
                if (r2 > rthr) { continue; }

                const float r_rc = rsqrtf(r2);
                float unit_rc_a1 = unit_a1 * r_rc;
                float t6 = unit_rc_a1 * unit_rc_a1; // ^2
                t6 *= unit_rc_a1; // ^3
                t6 *= t6; // ^6
                t6 *= unit_rc_a1; // ^7
                t6 *= t6; // ^14
                const float damp6 = 1.0f / fmaf(t6, 6.0f, 1.0f);
                float unit_rc_a2 = unit_a2 * r_rc;
                float t8 = unit_rc_a2 * unit_rc_a2; // ^2
                t8 *= t8; // ^4
                t8 *= t8; // ^8
                t8 *= t8; // ^16
                const float damp8 = 1.0f / fmaf(t8, 6.0f, 1.0f);
                const float r2_rc = r_rc * r_rc; // 1.0 / r2
                const float r6_rc = r2_rc * r2_rc * r2_rc;
                const float r8_rc = r6_rc * r2_rc;
                const float x1 = 6.0f * c6 * r8_rc * fmaf(r2_rc, s8r42 * damp8 * fmaf(3.0f * alp8 * t8, damp8, -4.0f), s6 * damp6 * fmaf(alp6 * t6, damp6, -1.0f));
                //const float x1 = 6.0 * c6 * r8_rc * (s6 * damp6 * (14.0 * t6 * damp6 - 1.0) + s8r42 * r2_rc * damp8 * (48.0 * t8 * damp8 - 4.0));
                //3.0 * alp6 = 48.0

                const float vec[3] = {
                    x1 * rij[0],
                    x1 * rij[1],
                    x1 * rij[2]
                };

                f_local[0] -= vec[0];
                f_local[1] -= vec[1];
                f_local[2] -= vec[2];

                sigma_local_00 += vec[0] * rij[0];
                sigma_local_01 += vec[0] * rij[1];
                sigma_local_02 += vec[0] * rij[2];
                sigma_local_10 += vec[1] * rij[0];
                sigma_local_11 += vec[1] * rij[1];
                sigma_local_12 += vec[1] * rij[2];
                sigma_local_20 += vec[2] * rij[0];
                sigma_local_21 += vec[2] * rij[1];
                sigma_local_22 += vec[2] * rij[2];

                const float dc6_rest = r6_rc * fmaf(3.0f * r2_rc, s8r42 * damp8, s6 * damp6);
                //const float dc6_rest = r6_rc * (s6 * damp6 + 3.0 * s8r42 * damp8 * r2_rc);
                disp_local -= dc6_rest * c6;
                dc6i_local_i += dc6_rest * dc6iji;
                dc6i_local_j += dc6_rest * dc6ijj;
            }
            atomicAdd(&dc6i[iat], dc6i_local_i);
            atomicAdd(&dc6i[jat], dc6i_local_j);
            atomicAdd(&f[iat][0], f_local[0]);
            atomicAdd(&f[iat][1], f_local[1]);
            atomicAdd(&f[iat][2], f_local[2]);
            atomicAdd(&f[jat][0], -f_local[0]);
            atomicAdd(&f[jat][1], -f_local[1]);
            atomicAdd(&f[jat][2], -f_local[2]);
        }
    }

    sigma_00[threadIdx.x] = sigma_local_00;
    sigma_01[threadIdx.x] = sigma_local_01;
    sigma_02[threadIdx.x] = sigma_local_02;
    sigma_10[threadIdx.x] = sigma_local_10;
    sigma_11[threadIdx.x] = sigma_local_11;
    sigma_12[threadIdx.x] = sigma_local_12;
    sigma_20[threadIdx.x] = sigma_local_20;
    sigma_21[threadIdx.x] = sigma_local_21;
    sigma_22[threadIdx.x] = sigma_local_22;
    disp_shared[threadIdx.x] = disp_local;
    __syncthreads();

    for (int s=blockDim.x/2; s>0; s>>=1) {
        if (threadIdx.x < s) {
            sigma_00[threadIdx.x] += sigma_00[threadIdx.x + s];
            sigma_01[threadIdx.x] += sigma_01[threadIdx.x + s];
            sigma_02[threadIdx.x] += sigma_02[threadIdx.x + s];
            sigma_10[threadIdx.x] += sigma_10[threadIdx.x + s];
            sigma_11[threadIdx.x] += sigma_11[threadIdx.x + s];
            sigma_12[threadIdx.x] += sigma_12[threadIdx.x + s];
            sigma_20[threadIdx.x] += sigma_20[threadIdx.x + s];
            sigma_21[threadIdx.x] += sigma_21[threadIdx.x + s];
            sigma_22[threadIdx.x] += sigma_22[threadIdx.x + s];
            disp_shared[threadIdx.x] += disp_shared[threadIdx.x + s];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        atomicAdd(&sigma[0][0], sigma_00[0]);
        atomicAdd(&sigma[0][1], sigma_01[0]);
        atomicAdd(&sigma[0][2], sigma_02[0]);
        atomicAdd(&sigma[1][0], sigma_10[0]);
        atomicAdd(&sigma[1][1], sigma_11[0]);
        atomicAdd(&sigma[1][2], sigma_12[0]);
        atomicAdd(&sigma[2][0], sigma_20[0]);
        atomicAdd(&sigma[2][1], sigma_21[0]);
        atomicAdd(&sigma[2][2], sigma_22[0]);
        atomicAdd(disp, disp_shared[0]);
    }
}

void PairD3::get_forces_without_dC6_zero() {
    int n = atom->natoms;
    int maxij = n * (n + 1) / 2;
    int maxtau = tau_idx_vdw_total_size;

    *disp = 0.0;

    for (int dim = 0; dim < n; dim++) { dc6i[dim] = 0.0; }

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < 3; j++) {
            f[i][j] = 0.0;
        }
    }

    for (int ii = 0; ii < 3; ii++) {
        for (int jj = 0; jj < 3; jj++) {
            sigma[ii][jj] = 0.0;
        }
    }

    //START_CUDA_TIMER();

    int threadsPerBlock = 128;
    int blocksPerGrid = (maxij + threadsPerBlock - 1) / threadsPerBlock;
    kernel_get_forces_without_dC6_zero<<<blocksPerGrid, threadsPerBlock>>>(
        maxij, maxtau, rthr, s6, s8, a1, a2, alp6, alp8,
        r2r4, r0ab, rep_vdw, tau_vdw, tau_idx_vdw, atomtype, x,
        c6_ij_tot, dc6_iji_tot, dc6_ijj_tot,
        dc6i, disp, f, sigma
    );
    hipDeviceSynchronize();
    disp_total = *disp;

    //STOP_CUDA_TIMER("get_forces_without");
}

__global__ void kernel_get_forces_without_dC6_bj(
    int maxij, int maxtau, float rthr, float s6, float s8, float a1, float a2,
    float *r2r4, int *rep_vdw, float ****tau_vdw, int *tau_idx_vdw, int *type, float **x,
    float *c6_ij_tot, float *dc6_iji_tot, float *dc6_ijj_tot,
    double *dc6i, double *disp, double **f, double **sigma
) {
    int iter = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ float sigma_00[128];
    __shared__ float sigma_01[128];
    __shared__ float sigma_02[128];
    __shared__ float sigma_10[128];
    __shared__ float sigma_11[128];
    __shared__ float sigma_12[128];
    __shared__ float sigma_20[128];
    __shared__ float sigma_21[128];
    __shared__ float sigma_22[128];
    __shared__ float disp_shared[128];

    float sigma_local_00 = 0.0f;
    float sigma_local_01 = 0.0f;
    float sigma_local_02 = 0.0f;
    float sigma_local_10 = 0.0f;
    float sigma_local_11 = 0.0f;
    float sigma_local_12 = 0.0f;
    float sigma_local_20 = 0.0f;
    float sigma_local_21 = 0.0f;
    float sigma_local_22 = 0.0f;
    float disp_local = 0.0f;

    if (iter < maxij) {
        int iat, jat;
        ij_at_linij(iter, iat, jat);

        float f_local[3] = { 0.0f };
        float dc6i_local_i = 0.0f;
        float dc6i_local_j = 0.0f;

        const float c6 = c6_ij_tot[iter];
        const float dc6iji = dc6_iji_tot[iter];
        const float dc6ijj = dc6_ijj_tot[iter];

        if (iat == jat) {
            const float unit_r2r4 = r2r4[type[iat]];
            const float r42x3 = unit_r2r4 * unit_r2r4 * 3.0f;
            const float R0 = fmaf(a1, sqrtf(r42x3), a2);
            const float R0_2 = R0 * R0;
            const float R0_6 = R0_2 * R0_2 * R0_2;
            const float R0_8 = R0_6 * R0_2;
            const float s8r42x3 = s8 * r42x3;

            for (int k = maxtau - 1; k >= 0; k -= 3) {
                const int idx1 = tau_idx_vdw[k-2];
                const int idx2 = tau_idx_vdw[k-1];
                const int idx3 = tau_idx_vdw[k];

                if (idx1 == rep_vdw[0] && idx2 == rep_vdw[1] && idx3 == rep_vdw[2]) { continue; }
                const float rij[3] = {
                    tau_vdw[idx1][idx2][idx3][0],
                    tau_vdw[idx1][idx2][idx3][1],
                    tau_vdw[idx1][idx2][idx3][2]
                };
                const float r2 = lensq3(rij);
                if (r2 > rthr) { continue; }

                const float r = sqrtf(r2);
                const float r5 = r2 * r2 * r;
                const float r7 = r5 * r2;
                const float t6_rc = 1.0f / fmaf(r5, r, R0_6);
                const float t8_rc = 1.0f / fmaf(r7, r, R0_8);
                const float t6_sqrc = t6_rc * t6_rc;
                const float t8_sqrc = t8_rc * t8_rc;
                const float x1 = -c6 * fmaf(4.0f * s8r42x3 * r7, t8_sqrc, 3.0f * s6 * r5 * t6_sqrc);
                //const float x1 = 0.5 * -c6 * (6.0 * s6 * r5 * t6_sqrc + 8.0 * s8r42x3 * r7 * t8_sqrc;

                const float r_rc = 1.0f / r; // rsqrt(r2)
                const float vec[3] = {
                    x1 * rij[0] * r_rc,
                    x1 * rij[1] * r_rc,
                    x1 * rij[2] * r_rc
                };

                sigma_local_00 += vec[0] * rij[0];
                sigma_local_01 += vec[0] * rij[1];
                sigma_local_02 += vec[0] * rij[2];
                sigma_local_10 += vec[1] * rij[0];
                sigma_local_11 += vec[1] * rij[1];
                sigma_local_12 += vec[1] * rij[2];
                sigma_local_20 += vec[2] * rij[0];
                sigma_local_21 += vec[2] * rij[1];
                sigma_local_22 += vec[2] * rij[2];

                const float dc6_rest = 0.5f * fmaf(s8r42x3, t8_rc, s6 * t6_rc);
                //const float dc6_rest = 0.5 * s6 * t6_rc + s8r42x3 * t8_rc;
                disp_local -= dc6_rest * c6;
                dc6i_local_i += dc6_rest * dc6iji;
                dc6i_local_j += dc6_rest * dc6ijj;
            }
            atomicAdd(&dc6i[iat], dc6i_local_i);
            atomicAdd(&dc6i[jat], dc6i_local_j);
        }

        else {
            const float r42x3 = r2r4[type[iat]] * r2r4[type[jat]] * 3.0f;
            const float R0 = fmaf(a1, sqrtf(r42x3), a2);
            const float R0_2 = R0 * R0;
            const float R0_6 = R0_2 * R0_2 * R0_2;
            const float R0_8 = R0_6 * R0_2;
            const float s8r42x3 = s8 * r42x3;

            for (int k = maxtau - 1; k >= 0; k -= 3) {
                const int idx1 = tau_idx_vdw[k-2];
                const int idx2 = tau_idx_vdw[k-1];
                const int idx3 = tau_idx_vdw[k];
                const float rij[3] = {
                    x[jat][0] - x[iat][0] + tau_vdw[idx1][idx2][idx3][0],
                    x[jat][1] - x[iat][1] + tau_vdw[idx1][idx2][idx3][1],
                    x[jat][2] - x[iat][2] + tau_vdw[idx1][idx2][idx3][2]
                };
                const float r2 = lensq3(rij);
                if (r2 > rthr) { continue; }

                const float r = sqrtf(r2);
                const float r5 = r2 * r2 * r;
                const float r7 = r5 * r2;
                const float t6_rc = 1.0f / fmaf(r5, r, R0_6);
                const float t8_rc = 1.0f / fmaf(r7, r, R0_8);
                const float t6_sqrc = t6_rc * t6_rc;
                const float t8_sqrc = t8_rc * t8_rc;
                const float x1 = -c6 * fmaf(8.0f * s8r42x3 * r7, t8_sqrc, 6.0f * s6 * r5 * t6_sqrc);
                //const float x1 = -c6 * (6.0 * s6 * r5 * t6_sqrc + 8.0 * s8r42x3 * r7 * t8_sqrc;

                const float r_rc = 1.0f / r; // rsqrt(r2)
                const float vec[3] = {
                    x1 * rij[0] * r_rc,
                    x1 * rij[1] * r_rc,
                    x1 * rij[2] * r_rc
                };

                f_local[0] -= vec[0];
                f_local[1] -= vec[1];
                f_local[2] -= vec[2];

                sigma_local_00 += vec[0] * rij[0];
                sigma_local_01 += vec[0] * rij[1];
                sigma_local_02 += vec[0] * rij[2];
                sigma_local_10 += vec[1] * rij[0];
                sigma_local_11 += vec[1] * rij[1];
                sigma_local_12 += vec[1] * rij[2];
                sigma_local_20 += vec[2] * rij[0];
                sigma_local_21 += vec[2] * rij[1];
                sigma_local_22 += vec[2] * rij[2];

                const float dc6_rest = fmaf(s8r42x3, t8_rc, s6 * t6_rc);
                //const float dc6_rest = s6 * t6_rc + s8r42x3 * t8_rc;
                disp_local -= dc6_rest * c6;
                dc6i_local_i += dc6_rest * dc6iji;
                dc6i_local_j += dc6_rest * dc6ijj;
            }
            atomicAdd(&dc6i[iat], dc6i_local_i);
            atomicAdd(&dc6i[jat], dc6i_local_j);
            atomicAdd(&f[iat][0], f_local[0]);
            atomicAdd(&f[iat][1], f_local[1]);
            atomicAdd(&f[iat][2], f_local[2]);
            atomicAdd(&f[jat][0], -f_local[0]);
            atomicAdd(&f[jat][1], -f_local[1]);
            atomicAdd(&f[jat][2], -f_local[2]);
        }
    }

    sigma_00[threadIdx.x] = sigma_local_00;
    sigma_01[threadIdx.x] = sigma_local_01;
    sigma_02[threadIdx.x] = sigma_local_02;
    sigma_10[threadIdx.x] = sigma_local_10;
    sigma_11[threadIdx.x] = sigma_local_11;
    sigma_12[threadIdx.x] = sigma_local_12;
    sigma_20[threadIdx.x] = sigma_local_20;
    sigma_21[threadIdx.x] = sigma_local_21;
    sigma_22[threadIdx.x] = sigma_local_22;
    disp_shared[threadIdx.x] = disp_local;
    __syncthreads();

    for (int s=blockDim.x/2; s>0; s>>=1) {
        if (threadIdx.x < s) {
            sigma_00[threadIdx.x] += sigma_00[threadIdx.x + s];
            sigma_01[threadIdx.x] += sigma_01[threadIdx.x + s];
            sigma_02[threadIdx.x] += sigma_02[threadIdx.x + s];
            sigma_10[threadIdx.x] += sigma_10[threadIdx.x + s];
            sigma_11[threadIdx.x] += sigma_11[threadIdx.x + s];
            sigma_12[threadIdx.x] += sigma_12[threadIdx.x + s];
            sigma_20[threadIdx.x] += sigma_20[threadIdx.x + s];
            sigma_21[threadIdx.x] += sigma_21[threadIdx.x + s];
            sigma_22[threadIdx.x] += sigma_22[threadIdx.x + s];
            disp_shared[threadIdx.x] += disp_shared[threadIdx.x + s];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        atomicAdd(&sigma[0][0], sigma_00[0]);
        atomicAdd(&sigma[0][1], sigma_01[0]);
        atomicAdd(&sigma[0][2], sigma_02[0]);
        atomicAdd(&sigma[1][0], sigma_10[0]);
        atomicAdd(&sigma[1][1], sigma_11[0]);
        atomicAdd(&sigma[1][2], sigma_12[0]);
        atomicAdd(&sigma[2][0], sigma_20[0]);
        atomicAdd(&sigma[2][1], sigma_21[0]);
        atomicAdd(&sigma[2][2], sigma_22[0]);
        atomicAdd(disp, disp_shared[0]);
    }
}

void PairD3::get_forces_without_dC6_bj() {
    int n = atom->natoms;
    int maxij = n * (n + 1) / 2;
    int maxtau = tau_idx_vdw_total_size;

    *disp = 0.0;

    for (int dim = 0; dim < n; dim++) { dc6i[dim] = 0.0; }

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < 3; j++) {
            f[i][j] = 0.0;
        }
    }

    for (int ii = 0; ii < 3; ii++) {
        for (int jj = 0; jj < 3; jj++) {
            sigma[ii][jj] = 0.0;
        }
    }

    //START_CUDA_TIMER();

    int threadsPerBlock = 128;
    int blocksPerGrid = (maxij + threadsPerBlock - 1) / threadsPerBlock;
    kernel_get_forces_without_dC6_bj<<<blocksPerGrid, threadsPerBlock>>>(
        maxij, maxtau, rthr, s6, s8, a1, a2,
        r2r4, rep_vdw, tau_vdw, tau_idx_vdw, atomtype, x,
        c6_ij_tot, dc6_iji_tot, dc6_ijj_tot,
        dc6i, disp, f, sigma
    );
    hipDeviceSynchronize();
    disp_total = *disp;

    //STOP_CUDA_TIMER("get_forces_without");
}

void PairD3::get_forces_without_dC6_zerom() {}
void PairD3::get_forces_without_dC6_bjm() {}

void PairD3::get_forces_without_dC6() {
    void (PairD3::*get_forces_without_dC6_damp[4])() = {
        &PairD3::get_forces_without_dC6_zero,
        &PairD3::get_forces_without_dC6_bj,
        &PairD3::get_forces_without_dC6_zerom,
        &PairD3::get_forces_without_dC6_bjm
    };

    (this->*get_forces_without_dC6_damp[damping])();
}

__global__ void kernel_get_forces_with_dC6(
    int maxij, int maxtau, float cnthr, float K1,
    double *dc6i, float *rcov, int *rep_cn, float ****tau_cn, int *tau_idx_cn, int *type, float **x,
    double **f, double **sigma
) {
    int iter = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ float sigma_00[128];
    __shared__ float sigma_01[128];
    __shared__ float sigma_02[128];
    __shared__ float sigma_10[128];
    __shared__ float sigma_11[128];
    __shared__ float sigma_12[128];
    __shared__ float sigma_20[128];
    __shared__ float sigma_21[128];
    __shared__ float sigma_22[128];

    float sigma_local_00 = 0.0f;
    float sigma_local_01 = 0.0f;
    float sigma_local_02 = 0.0f;
    float sigma_local_10 = 0.0f;
    float sigma_local_11 = 0.0f;
    float sigma_local_12 = 0.0f;
    float sigma_local_20 = 0.0f;
    float sigma_local_21 = 0.0f;
    float sigma_local_22 = 0.0f;

    float f_local[3] = { 0.0f };

    if (iter < maxij) {
        int iat, jat;
        ij_at_linij(iter, iat, jat);

        if (iat == jat) {
            const float rcov_sum = rcov[type[iat]] * 2.0f;
            const float dc6i_sum = dc6i[iat];

            for (int k = maxtau - 1; k >= 0; k -= 3) {
                const int idx1 = tau_idx_cn[k-2];
                const int idx2 = tau_idx_cn[k-1];
                const int idx3 = tau_idx_cn[k];

                if (idx1 == rep_cn[0] && idx2 == rep_cn[1] && idx3 == rep_cn[2]) { continue; }
                const float rij[3] = {
                    tau_cn[idx1][idx2][idx3][0],
                    tau_cn[idx1][idx2][idx3][1],
                    tau_cn[idx1][idx2][idx3][2],
                };
                const float r2 = lensq3(rij);
                if (r2 >= cnthr) { continue; }

                const float r_rc = rsqrtf(r2);
                const float expterm = expf(-K1 * (rcov_sum * r_rc - 1.0f));
                const float unit_rc = 1.0f / (r2 * (expterm + 1.0f) * (expterm + 1.0f));
                const float dcnn = -K1 * rcov_sum * expterm * unit_rc;
                const float x1 = dcnn * dc6i_sum;

                const float vec[3] = {
                    x1 * rij[0] * r_rc,
                    x1 * rij[1] * r_rc,
                    x1 * rij[2] * r_rc
                };

                sigma_local_00 += vec[0] * rij[0];
                sigma_local_01 += vec[0] * rij[1];
                sigma_local_02 += vec[0] * rij[2];
                sigma_local_10 += vec[1] * rij[0];
                sigma_local_11 += vec[1] * rij[1];
                sigma_local_12 += vec[1] * rij[2];
                sigma_local_20 += vec[2] * rij[0];
                sigma_local_21 += vec[2] * rij[1];
                sigma_local_22 += vec[2] * rij[2];
            }
        }

        else {
            const float rcov_sum = rcov[type[iat]] + rcov[type[jat]];
            const float dc6i_sum = dc6i[iat] + dc6i[jat];

            for (int k = maxtau - 1; k >= 0; k -= 3) {
                const int idx1 = tau_idx_cn[k-2];
                const int idx2 = tau_idx_cn[k-1];
                const int idx3 = tau_idx_cn[k];

                const float rij[3] = {
                    x[jat][0] - x[iat][0] + tau_cn[idx1][idx2][idx3][0],
                    x[jat][1] - x[iat][1] + tau_cn[idx1][idx2][idx3][1],
                    x[jat][2] - x[iat][2] + tau_cn[idx1][idx2][idx3][2]
                };
                const float r2 = lensq3(rij);
                if (r2 >= cnthr) { continue; }

                const float r_rc = rsqrtf(r2);
                const float expterm = expf(-K1 * (rcov_sum * r_rc - 1.0f));
                const float unit_rc = 1.0f / (r2 * (expterm + 1.0f) * (expterm + 1.0f));
                const float dcnn = -K1 * rcov_sum * expterm * unit_rc;
                const float x1 = dcnn * dc6i_sum;

                const float vec[3] = {
                    x1 * rij[0] * r_rc,
                    x1 * rij[1] * r_rc,
                    x1 * rij[2] * r_rc
                };

                f_local[0] -= vec[0];
                f_local[1] -= vec[1];
                f_local[2] -= vec[2];

                sigma_local_00 += vec[0] * rij[0];
                sigma_local_01 += vec[0] * rij[1];
                sigma_local_02 += vec[0] * rij[2];
                sigma_local_10 += vec[1] * rij[0];
                sigma_local_11 += vec[1] * rij[1];
                sigma_local_12 += vec[1] * rij[2];
                sigma_local_20 += vec[2] * rij[0];
                sigma_local_21 += vec[2] * rij[1];
                sigma_local_22 += vec[2] * rij[2];
            }
            atomicAdd(&f[iat][0], f_local[0]);
            atomicAdd(&f[iat][1], f_local[1]);
            atomicAdd(&f[iat][2], f_local[2]);
            atomicAdd(&f[jat][0], -f_local[0]);
            atomicAdd(&f[jat][1], -f_local[1]);
            atomicAdd(&f[jat][2], -f_local[2]);
        }
    }

    sigma_00[threadIdx.x] = sigma_local_00;
    sigma_01[threadIdx.x] = sigma_local_01;
    sigma_02[threadIdx.x] = sigma_local_02;
    sigma_10[threadIdx.x] = sigma_local_10;
    sigma_11[threadIdx.x] = sigma_local_11;
    sigma_12[threadIdx.x] = sigma_local_12;
    sigma_20[threadIdx.x] = sigma_local_20;
    sigma_21[threadIdx.x] = sigma_local_21;
    sigma_22[threadIdx.x] = sigma_local_22;
    __syncthreads();

    for (int s=blockDim.x/2; s>0; s>>=1) {
        if (threadIdx.x < s) {
            sigma_00[threadIdx.x] += sigma_00[threadIdx.x + s];
            sigma_01[threadIdx.x] += sigma_01[threadIdx.x + s];
            sigma_02[threadIdx.x] += sigma_02[threadIdx.x + s];
            sigma_10[threadIdx.x] += sigma_10[threadIdx.x + s];
            sigma_11[threadIdx.x] += sigma_11[threadIdx.x + s];
            sigma_12[threadIdx.x] += sigma_12[threadIdx.x + s];
            sigma_20[threadIdx.x] += sigma_20[threadIdx.x + s];
            sigma_21[threadIdx.x] += sigma_21[threadIdx.x + s];
            sigma_22[threadIdx.x] += sigma_22[threadIdx.x + s];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        atomicAdd(&sigma[0][0], sigma_00[0]);
        atomicAdd(&sigma[0][1], sigma_01[0]);
        atomicAdd(&sigma[0][2], sigma_02[0]);
        atomicAdd(&sigma[1][0], sigma_10[0]);
        atomicAdd(&sigma[1][1], sigma_11[0]);
        atomicAdd(&sigma[1][2], sigma_12[0]);
        atomicAdd(&sigma[2][0], sigma_20[0]);
        atomicAdd(&sigma[2][1], sigma_21[0]);
        atomicAdd(&sigma[2][2], sigma_22[0]);
    }
}

void PairD3::get_forces_with_dC6() {
    int n = atom->natoms;
    int maxij = n * (n + 1) / 2;
    int maxtau = tau_idx_cn_total_size;

    //START_CUDA_TIMER();

    int threadsPerBlock = 128;
    int blocksPerGrid = (maxij + threadsPerBlock - 1) / threadsPerBlock;
    kernel_get_forces_with_dC6<<<blocksPerGrid, threadsPerBlock>>>(
        maxij, maxtau, cnthr, K1,
        dc6i, rcov, rep_cn, tau_cn, tau_idx_cn, atomtype, x,
        f, sigma
    );
    hipDeviceSynchronize();

    //STOP_CUDA_TIMER("get_forces_with");
}

/* ----------------------------------------------------------------------
   Update energy, force, and stress
------------------------------------------------------------------------- */

void PairD3::update(int eflag, int vflag) {
    int n = atom->natoms;

    if (eflag) { eng_vdwl += disp_total * AU_TO_EV; } // Energy update

    double** f_local = atom->f; // Force update
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < 3; j++) {
            f_local[i][j] += f[i][j] * AU_TO_EV / AU_TO_ANG;
        }
    }


    if (vflag) {
        virial[0] += sigma[0][0] * AU_TO_EV;
        virial[1] += sigma[1][1] * AU_TO_EV;
        virial[2] += sigma[2][2] * AU_TO_EV;
        virial[3] += sigma[0][1] * AU_TO_EV;
        virial[4] += sigma[0][2] * AU_TO_EV;
        virial[5] += sigma[1][2] * AU_TO_EV;
    } // Stress update
}

/* ----------------------------------------------------------------------
   Compute : energy, force, and stress (Required)
------------------------------------------------------------------------- */

void PairD3::compute(int eflag, int vflag) {
    if (eflag || vflag)          { ev_setup(eflag, vflag); }
    if (atom->natoms != n_save)  { reallocate_arrays(); }

    set_lattice_vectors();
    precalculate_tau_array();
    load_atom_info();

    hipMemcpy(atomtype, atom->type, atom->natoms * sizeof(int), hipMemcpyHostToDevice);

    get_coordination_number();
    get_dC6_dCNij();
    get_forces_without_dC6();
    get_forces_with_dC6();

    update(eflag, vflag);

    CHECK_CUDA_ERROR();
}

/* ----------------------------------------------------------------------
   init for one type pair i,j and corresponding j,i
------------------------------------------------------------------------- */

double PairD3::init_one(int i, int j) {
    if (setflag[i][j] == 0) error->all(FLERR, "All pair coeffs are not set");
    // No need to count local neighbor in D3
    /* return std::sqrt(rthr * std::pow(au_to_ang, 2)); */
    return 0.0;
}

/* ----------------------------------------------------------------------
   init specific to this pair style (Optional)
------------------------------------------------------------------------- */

void PairD3::init_style() {
    neighbor->add_request(this, NeighConst::REQ_FULL);
}

/* ----------------------------------------------------------------------
   proc 0 writes to restart file
------------------------------------------------------------------------- */

void PairD3::write_restart(FILE *fp) {}

/* ----------------------------------------------------------------------
   proc 0 reads from restart file, bcasts
------------------------------------------------------------------------- */

void PairD3::read_restart(FILE *fp) {}

/* ----------------------------------------------------------------------
   proc 0 writes to restart file
------------------------------------------------------------------------- */

void PairD3::write_restart_settings(FILE *fp) {}

/* ----------------------------------------------------------------------
   proc 0 reads from restart file, bcasts
------------------------------------------------------------------------- */

void PairD3::read_restart_settings(FILE *fp) {}
